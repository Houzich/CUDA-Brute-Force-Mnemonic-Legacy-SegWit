﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.2.0
  * @date		16-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"






int Generate_Mnemonic(void)
{
	hipError_t cudaStatus = hipSuccess;

	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}

	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //TEST_MODE
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}
	//18,446,744,073,709,551,615
	size_t number_of_addresses = 0;
	size_t count_save_data_in_file = 0;
	int num_bytes = 0;
	int err;
	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
#ifndef TEST_MODE
	std::cout << "Max value: 18,000,000,000,000,000,000 (18000000000000000000)" << std::endl;
	std::cout << "Enter number of generate mnemonic: ";
	std::cin >> number_of_addresses;
	number_of_addresses = (((number_of_addresses - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));

	std::cout << "Enter num rounds save data in file: ";
	std::cin >> count_save_data_in_file;

	std::cout << "Enter num bytes for check 6...8: ";
	std::cin >> num_bytes;
	if (num_bytes != 0)
		if ((num_bytes < 6) || (num_bytes > 8)) {
			std::cout << "Error num bytes. Won't be used!" << std::endl;
			num_bytes = 0;
		}


#else
	//number_of_addresses = 18 000 000 000 000 000 000;
	number_of_addresses = ((((num_wallets_gpu * 1) - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));
	//bip44_test_str = "";
	num_bytes = 5;
	count_save_data_in_file = 2;
#endif //TEST_MODE

	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	if((Config.generate_path[0] != 0) || (Config.generate_path[1] != 0) || (Config.generate_path[2] != 0) || (Config.generate_path[3] != 0) || (Config.generate_path[4] != 0)
		|| (Config.generate_path[5] != 0))
	{
		std::cout << "READ TABLES LEGACY(BIP32, BIP44)..." << std::endl;
	err = tools::readAllTables(Data->host.tables_legacy, Config.folder_tables_legacy, "");
	if (err == -1) {
		std::cout << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	}
	if ((Config.generate_path[6] != 0) || (Config.generate_path[7] != 0))
	{
		std::cout << "READ TABLES SEGWIT(BIP49)..." << std::endl;
		err = tools::readAllTables(Data->host.tables_segwit, Config.folder_tables_segwit, "");
		if (err == -1) {
			std::cout << "Error readAllTables segwit!" << std::endl;
			goto Error;
		}
	}
	if ((Config.generate_path[8] != 0) || (Config.generate_path[9] != 0))
	{
		std::cout << "READ TABLES NATIVE SEGWIT(BIP84)..." << std::endl;
		err = tools::readAllTables(Data->host.tables_native_segwit, Config.folder_tables_native_segwit, "");
		if (err == -1) {
			std::cout << "Error readAllTables native segwit!" << std::endl;
			goto Error;
		}
	}
	std::cout << std::endl << std::endl;

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, count_save_data_in_file == 0 ? false : true) != 0) {
		std::cout << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		printf("Error INIT!!\n");
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;
	if (Config.generate_path[0] != 0) std::cout << "m/0/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[1] != 0) std::cout << "m/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[2] != 0) std::cout << "m/0/0/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[3] != 0) std::cout << "m/0/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[4] != 0) std::cout << "m/44'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[5] != 0) std::cout << "m/44'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[6] != 0) std::cout << "m/49'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[7] != 0) std::cout << "m/49'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[8] != 0) std::cout << "m/84'/0'/0'/0/0.." << (Config.num_child_addresses - 1) << std::endl;
	if (Config.generate_path[9] != 0) std::cout << "m/84'/0'/0'/1/0.." << (Config.num_child_addresses - 1) << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(number_of_addresses) << " MNEMONICS. " << tools::formatWithCommas(number_of_addresses * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_bytes_find failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to dev_generate_path failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_child), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to dev_num_child failed!");
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to dev_num_paths failed!");
		goto Error;
	}
	static int start_save = 0;
	for (uint64_t step = 0; step < number_of_addresses / (Data->wallets_in_round_gpu); step++)
	{
		tools::start_time();

		if (start_save < count_save_data_in_file) {
			if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}


		tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
		if (start_save < count_save_data_in_file) {
			if (Stride->end_for_save() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}

		if (start_save < count_save_data_in_file) {
			start_save++;
			tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs, Data->num_childs, Config.generate_path);
		}

		tools::checkResult(Data->host.ret);

		float delay;
		tools::stop_time_and_calc(&delay);
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)Data->wallets_in_round_gpu / (delay / 1000.0f)) << " MNEMONICS/SECOND AND "
			<< tools::formatWithCommas(((float)Data->wallets_in_round_gpu * Data->num_all_childs) / (delay / 1000.0f)) << " ADDRESSES/SECOND, ROUND: " << step;

	}
	std::cout << "\n\nEND!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







