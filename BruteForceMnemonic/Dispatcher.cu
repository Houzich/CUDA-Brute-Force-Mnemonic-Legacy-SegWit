﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.0.0
  * @date		20-March-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"



uint64_t number_of_addresses_generate = 0;
int num_bytes = 0;




int Generate_Mnemonic_And_Hash(void)
{
	hipError_t cudaStatus = hipSuccess;

	ConfigClass config;
	parse_gonfig(&config, "config.cfg");

	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef GENERATE_INFINITY
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = config.cuda_grid * config.cuda_block;

	tools::Clear_Files();
	//18,446,744,073,709,551,615
	uint64_t number_of_addresses = 0;
	int count_save_data_in_file = 0;

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
#ifndef GENERATE_INFINITY
	std::cout << "Max value: 18,000,000,000,000,000,000 (18000000000000000000)" << std::endl;
	std::cout << "Enter number of seeds: ";
	std::cin >> number_of_addresses;
	number_of_addresses = (((number_of_addresses - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));

	std::cout << "Enter num cycles save data in file: ";
	std::cin >> count_save_data_in_file;

	std::cout << "!!!FOR TEST!!! Enter num bytes for check 5...8: ";
	std::cin >> num_bytes;
	if (num_bytes != 0)
		if ((num_bytes < 5) || (num_bytes > 8)) {
			std::cout << "Error num bytes. Won't be used!" << std::endl;
			num_bytes = 0;
		}


#else
	//number_of_addresses = 18 000 000 000 000 000 000;
	number_of_addresses = ((((num_wallets_gpu * 10) - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));
	//bip44_test_str = "";
	num_bytes = 0;
	count_save_data_in_file = 0;
#endif //GENERATE_INFINITY

	data_class* Board = new data_class();
	stride_class* Stride = new stride_class(Board);

#ifdef GENERATE_SEGWIT
	int err = tools::get_all_tables(Board->host.tables_segwit, config.folder_database_segwit, config.prefix_database_segwit);
	if (err == -1) {
		std::cout << "Error get_all_tables segwit!" << std::endl;
		goto Error;
	}
#elif defined (GENERATE_LEGACY_AND_SEGWIT)
	int err = tools::get_all_tables(Board->host.tables_legacy, config.folder_database_legacy, config.prefix_database_legacy);
	if (err == -1) {
		std::cout << "Error get_all_tables legacy!" << std::endl;
		goto Error;
	}
	err = tools::get_all_tables(Board->host.tables_segwit, config.folder_database_segwit, config.prefix_database_segwit);
	if (err == -1) {
		std::cout << "Error get_all_tables segwit!" << std::endl;
		goto Error;
	}
#endif //GENERATE_BIP32


	if (Board->Malloc(config.cuda_grid, config.cuda_block, count_save_data_in_file == 0 ? false : true) != 0) {
		std::cout << "Error Board->Malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		printf("Error INIT!!\n");
		goto Error;
	}

	Board->host.free_table_buffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(number_of_addresses) << " SEDDS. " << tools::formatWithCommas(number_of_addresses * NUM_ALL_CHILDS) << " CHILD ADDRESSES. PACKET " << tools::formatWithCommas(Board->num_wallets_gpu) << ". WAIT...\n\n";

	tools::Generate_Random_LongWords_Byffer(Board->host.entropy, Board->size_entropy_buf / (sizeof(uint64_t)));


	//for (uint64_t i = 0; i < 256; i++)
	//{
	//	if (hipMemcpyToSymbol(HIP_SYMBOL(bip84_tables[i]), Board->host.table[i], Board->host.table_size[i], 0, hipMemcpyHostToDevice) != hipSuccess)
	//	{
	//		fprintf(stderr, "hipMemcpyToSymbol to tables[256] failed! i = %d", i);
	//		goto Error;
	//	}
	//}

	//std::cout << "tables_pionts size: " << sizeof(host_buffers_class::tables_points) << std::endl;

	//if (hipMemcpyToSymbol(HIP_SYMBOL(bip84_table_size), Board->host.table_size, sizeof(host_buffers_class::table_size), 0, hipMemcpyHostToDevice) != hipSuccess)
	//{
	//	fprintf(stderr, "hipMemcpyToSymbol to table_size[256] failed!");
	//	goto Error;
	//}




	if (hipMemcpyToSymbol(HIP_SYMBOL(num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_bytes_find failed!");
		goto Error;
	}


	static int start_save = 0;
	for (uint64_t step = 0; step < number_of_addresses / (Board->num_wallets_gpu); step++)
	{
		tools::start_time();

		number_of_addresses_generate = (step + 1) * (Board->num_wallets_gpu);
		if (start_save < count_save_data_in_file) {
			if (Stride->start_for_save(config.cuda_grid, config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->start(config.cuda_grid, config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}


		tools::Generate_Random_LongWords_Byffer(Board->host.entropy, Board->size_entropy_buf / (sizeof(uint64_t)));
		if (start_save < count_save_data_in_file) {
			if (Stride->end_for_save() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}

		if (start_save < count_save_data_in_file) {
			start_save++;
			tools::Save_Result((char*)Board->host.mnemonic, (uint8_t*)Board->host.hash160, Board->num_wallets_gpu);
		}

		tools::Print_Save_Ret(Board->host.ret);

		float delay;
		tools::stop_time_and_calc(&delay);
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)Board->num_wallets_gpu / (delay / 1000.0f)) << " SEEDS AND "
			<< tools::formatWithCommas(((float)Board->num_wallets_gpu * NUM_ALL_CHILDS) / (delay / 1000.0f)) << " ADDRESSES PER SECOND, ROUND: " << step;

	}
	std::cout << "\n\nEND!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







