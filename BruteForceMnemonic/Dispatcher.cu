﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.0.0
  * @date		20-March-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"






int Generate_Mnemonic(void)
{
	hipError_t cudaStatus = hipSuccess;

	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}

	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef GENERATE_INFINITY
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	//18,446,744,073,709,551,615
	size_t number_of_addresses = 0;
	size_t count_save_data_in_file = 0;
	int num_bytes = 0;

	std::cout << "\nNUM WALLETS IN PACKET GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
#ifndef GENERATE_INFINITY
	std::cout << "Max value: 18,000,000,000,000,000,000 (18000000000000000000)" << std::endl;
	std::cout << "Enter number of generate mnemonic: ";
	std::cin >> number_of_addresses;
	number_of_addresses = (((number_of_addresses - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));

	std::cout << "Enter num rounds save data in file: ";
	std::cin >> count_save_data_in_file;

	std::cout << "Enter num bytes for check 6...8: ";
	std::cin >> num_bytes;
	if (num_bytes != 0)
		if ((num_bytes < 6) || (num_bytes > 8)) {
			std::cout << "Error num bytes. Won't be used!" << std::endl;
			num_bytes = 0;
		}


#else
	//number_of_addresses = 18 000 000 000 000 000 000;
	number_of_addresses = ((((num_wallets_gpu * 10) - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));
	//bip44_test_str = "";
	num_bytes = 0;
	count_save_data_in_file = 0;
#endif //GENERATE_INFINITY

	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
#ifdef GENERATE_SEGWIT
	int err = tools::readAllTables(Board->host.tables_segwit, Config.folder_database_segwit, "");
	if (err == -1) {
		std::cout << "Error readAllTables segwit!" << std::endl;
		goto Error;
	}
#elif defined (GENERATE_LEGACY_AND_SEGWIT)
	int err = tools::readAllTables(Data->host.tables_legacy, Config.folder_database_legacy, "");
	if (err == -1) {
		std::cout << "Error readAllTables legacy!" << std::endl;
		goto Error;
	}
	err = tools::readAllTables(Data->host.tables_segwit, Config.folder_database_segwit, "");
	if (err == -1) {
		std::cout << "Error readAllTables segwit!" << std::endl;
		goto Error;
	}
#endif //GENERATE_BIP32

	std::cout << std::endl << std::endl;

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, count_save_data_in_file == 0 ? false : true) != 0) {
		std::cout << "Error Data->malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		printf("Error INIT!!\n");
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "PATH: " << std::endl;
	std::cout << "m/0/0.." << (NUM_CHILDS - 1) << ", m/1/0.." << (NUM_CHILDS - 1) << std::endl;
	std::cout << "m/0/0/0.." << (NUM_CHILDS - 1) << ", m/0/1/0.." << (NUM_CHILDS - 1) << std::endl;
	std::cout << "m/44'/0'/0'/0/0.." << (NUM_CHILDS - 1) << ", m/44'/0'/0'/1/0.." << (NUM_CHILDS - 1) << std::endl;
	std::cout << "m/84'/0'/0'/0/0.." << (NUM_CHILDS - 1) << ", m/84'/0'/0'/1/0.." << (NUM_CHILDS - 1) << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(number_of_addresses) << " MNEMONICS. " << tools::formatWithCommas(number_of_addresses * NUM_ALL_CHILDS) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";

	tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));

	if (hipMemcpyToSymbol(HIP_SYMBOL(num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_bytes_find failed!");
		goto Error;
	}


	static int start_save = 0;
	for (uint64_t step = 0; step < number_of_addresses / (Data->wallets_in_round_gpu); step++)
	{
		tools::start_time();

		if (start_save < count_save_data_in_file) {
			if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}


		tools::generateRandomUint64Buffer(Data->host.entropy, Data->size_entropy_buf / (sizeof(uint64_t)));
		if (start_save < count_save_data_in_file) {
			if (Stride->end_for_save() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}

		if (start_save < count_save_data_in_file) {
			start_save++;
			tools::saveResult((char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu);
		}

		tools::checkResult(Data->host.ret);

		float delay;
		tools::stop_time_and_calc(&delay);
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)Data->wallets_in_round_gpu / (delay / 1000.0f)) << " MNEMONICS/SECOND AND "
			<< tools::formatWithCommas(((float)Data->wallets_in_round_gpu * NUM_ALL_CHILDS) / (delay / 1000.0f)) << " ADDRESSES/SECOND, ROUND: " << step;

	}
	std::cout << "\n\nEND!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







