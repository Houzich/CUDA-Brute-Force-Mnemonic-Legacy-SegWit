﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.0.0
  * @date		20-March-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>
#include <stdio.h>
#include <stdint.h>


#include "KernelStride.hpp"
#include "Helper.h"
#include <GPU.h>
#include "../Tools/utils.h"


int stride_class::bruteforce_mnemonic(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables_legacy, dt->dev.dev_tables_segwit, dt->dev.ret);
	//______________________________________________________________________________________________________________
	//gl_test_1 << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (
	//	dt->dev.entropy,
	//	dt->dev.mnemonic_buff,
	//	dt->dev.ipad_buff,
	//	dt->dev.opad_buff,
	//	dt->dev.seed_buff
	//	);
	//if (DeviceSynchronize("gl_test_1") != hipSuccess) return -1;
	//for (int x = 1; x < 2048; x++)
	//gl_test_2 << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (
	//	dt->dev.ipad_buff,
	//	dt->dev.opad_buff,
	//	dt->dev.seed_buff
	//	);
	//if (DeviceSynchronize("gl_test_2") != hipSuccess) return -1;
	//gl_test_3 << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (
	//	dt->dev.dev_tables_legacy,
	//	dt->dev.dev_tables_segwit,
	//	dt->dev.mnemonic_buff,
	//	dt->dev.ipad_buff,
	//	dt->dev.opad_buff,
	//	dt->dev.seed_buff,
	//	dt->dev.ret
	//	);
	//if (DeviceSynchronize("gl_test_3") != hipSuccess) return -1;
	//______________________________________________________________________________________________________________
	return 0;
}

int stride_class::bruteforce_mnemonic_for_save(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic_for_save << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables_legacy, dt->dev.dev_tables_segwit, dt->dev.ret, dt->dev.mnemonic, dt->dev.hash160);
	return 0;
}

int stride_class::memsetGlobalMnemonic()
{
	//if (DeviceSynchronize("hipMemcpy table") != hipSuccess) return -1;
	if (hipMemcpyAsync(dt->dev.entropy, dt->host.entropy, dt->size_entropy_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.entropy failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	//______________________________________________________________________________________________________________
	//if (hipMemsetAsync(dt->dev.mnemonic_buff, 0, dt->num_wallets_gpu * SIZE_MNEMONIC_FRAME, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.mnemonic_buff failed!"); return -1; }
	//if (hipMemsetAsync(dt->dev.seed_buff, 0, dt->num_wallets_gpu * 64, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.seed_buff failed!"); return -1; }
	//______________________________________________________________________________________________________________
	return 0;
}

int stride_class::memsetGlobalMnemonicSave()
{
	if (hipMemcpyAsync(dt->dev.entropy, dt->host.entropy, dt->size_entropy_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.entropy failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int stride_class::init()
{
	size_t memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (dt->dev.cudaMallocDevice((uint8_t**)&dt->dev.tables_legacy[i].table, dt->host.tables_legacy[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table_legacy[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables_legacy[i].size = dt->host.tables_legacy[i].size;
		dt->dev.memory_size += dt->host.tables_legacy[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES LEGACY GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";
	memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (dt->dev.cudaMallocDevice((uint8_t**)&dt->dev.tables_segwit[i].table, dt->host.tables_segwit[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table_segwit[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables_segwit[i].size = dt->host.tables_segwit[i].size;
		dt->dev.memory_size += dt->host.tables_segwit[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES SEGWIT GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";


	std::cout << "INIT GPU ... \n";
	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables_legacy[i].table, dt->host.tables_legacy[i].table, dt->host.tables_legacy[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table_legacy[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = (i * 100 / 256) / 2;
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables_legacy, dt->dev.tables_legacy, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table_legacy failed!"); return -1; }

	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables_segwit[i].table, dt->host.tables_segwit[i].table, dt->host.tables_segwit[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table_segwit[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = 50 + (i * 100 / 256) / 2;
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables_segwit, dt->dev.tables_segwit, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table_segwit failed!"); return -1; }
	if (deviceSynchronize("init") != hipSuccess) return -1;
	return 0;
}

int stride_class::start_for_save(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonicSave() != 0) return -1;
	if (bruteforce_mnemonic_for_save(grid, block) != 0) return -1;

	return 0;
}

int stride_class::start(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonic() != 0) return -1;
	if (bruteforce_mnemonic(grid, block) != 0) return -1;

	return 0;
}

int stride_class::end()
{
	hipError_t cudaStatus = hipSuccess;
	if (deviceSynchronize("end") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}

int stride_class::end_for_save()
{
	hipError_t cudaStatus = hipSuccess;


	if (deviceSynchronize("end_for_save") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.mnemonic, dt->dev.mnemonic, dt->size_mnemonic_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy mnemonic failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.hash160, dt->dev.hash160, dt->size_hash160_bip44_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy hash160 failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}