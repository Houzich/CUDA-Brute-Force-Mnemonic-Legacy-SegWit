﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.2.0
  * @date		16-April-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>
#include <stdio.h>
#include <stdint.h>


#include "KernelStride.hpp"
#include "Helper.h"
#include <GPU.h>
#include "../Tools/utils.h"


int stride_class::bruteforce_mnemonic(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables_legacy, dt->dev.dev_tables_segwit, dt->dev.dev_tables_native_segwit, dt->dev.ret);
	return 0;
}

int stride_class::bruteforce_mnemonic_for_save(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic_for_save << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.entropy, dt->dev.dev_tables_legacy, dt->dev.dev_tables_segwit, dt->dev.dev_tables_native_segwit, dt->dev.ret, dt->dev.mnemonic, dt->dev.hash160);
	return 0;
}

int stride_class::memsetGlobalMnemonic()
{
	//if (DeviceSynchronize("hipMemcpy table") != hipSuccess) return -1;
	if (hipMemcpyAsync(dt->dev.entropy, dt->host.entropy, dt->size_entropy_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.entropy failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int stride_class::memsetGlobalMnemonicSave()
{
	if (hipMemcpyAsync(dt->dev.entropy, dt->host.entropy, dt->size_entropy_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.entropy failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int stride_class::init()
{
	size_t memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (dt->dev.cudaMallocDevice((uint8_t**)&dt->dev.tables_legacy[i].table, dt->host.tables_legacy[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table_legacy[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables_legacy[i].size = dt->host.tables_legacy[i].size;
		dt->dev.memory_size += dt->host.tables_legacy[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES LEGACY(BIP32, BIP44) GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";

	memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (dt->dev.cudaMallocDevice((uint8_t**)&dt->dev.tables_segwit[i].table, dt->host.tables_segwit[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.tables_segwit[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables_segwit[i].size = dt->host.tables_segwit[i].size;
		dt->dev.memory_size += dt->host.tables_segwit[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES SEGWIT(BIP49) GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";

	memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (dt->dev.cudaMallocDevice((uint8_t**)&dt->dev.tables_native_segwit[i].table, dt->host.tables_native_segwit[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.tables_native_segwit[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables_native_segwit[i].size = dt->host.tables_native_segwit[i].size;
		dt->dev.memory_size += dt->host.tables_native_segwit[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES NATIVE SEGWIT(BIP84) GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";


	std::cout << "INIT GPU ... \n";
	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables_legacy[i].table, dt->host.tables_legacy[i].table, dt->host.tables_legacy[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table_legacy[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = (i * 100 / 256) / 2;
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables_legacy, dt->dev.tables_legacy, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table_legacy failed!"); return -1; }

	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables_native_segwit[i].table, dt->host.tables_native_segwit[i].table, dt->host.tables_native_segwit[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table_segwit[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = 50 + (i * 100 / 256) / 2;
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables_native_segwit, dt->dev.tables_native_segwit, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table_segwit failed!"); return -1; }
	if (deviceSynchronize("init") != hipSuccess) return -1;
	return 0;
}

int stride_class::start_for_save(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonicSave() != 0) return -1;
	if (bruteforce_mnemonic_for_save(grid, block) != 0) return -1;

	return 0;
}

int stride_class::start(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonic() != 0) return -1;
	if (bruteforce_mnemonic(grid, block) != 0) return -1;

	return 0;
}

int stride_class::end()
{
	hipError_t cudaStatus = hipSuccess;
	if (deviceSynchronize("end") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}

int stride_class::end_for_save()
{
	hipError_t cudaStatus = hipSuccess;


	if (deviceSynchronize("end_for_save") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.mnemonic, dt->dev.mnemonic, dt->size_mnemonic_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy mnemonic failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.hash160, dt->dev.hash160, dt->size_hash160_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy hash160 failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}